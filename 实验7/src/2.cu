
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>

__global__ void im2colOnDevice(unsigned int n, float *matAc, float *matA, int radiusF, int countF, int L, int M, int K, int C,int H)
{
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n; idx += blockDim.x * gridDim.x) {
        int m = (idx / C) / L;
        int l = (idx / C) % L;
        int r = idx % C;
        if (m < M) {
            int w = m + radiusF;
            if (l < L) {
                int h = l + radiusF;
                for (int q = 0, oq = -1 * radiusF; oq <= radiusF; q++, oq++) {
                    for (int p = 0, op = -1 * radiusF; op <= radiusF; p++, op++) {
                        if (r < C) {
                            matAc[(r + C * (p + K * q)) + countF * (l + L * m)] = matA[r + C * ((h + op) + H * (w + oq))]; 
                        }
                    }
                }
            }
        }
    }
}

__global__ void gemm_gpu(double *a,double *b, double *c, int m, int n, int k)
{ 
   int row = blockIdx.y * blockDim.y + threadIdx.y; 
   int col = blockIdx.x * blockDim.x + threadIdx.x;
   double tmp = 0;
   if( col < k && row < m) {
       for(int i = 0; i < n; i++) {
           tmp += a[row * n + i] * b[i * k + col];
       }
       c[row * k + col] = tmp;
   }
}

int main(int argc, char const *argv[])
{
    int W=atoi(argv[1]);
    int H=atoi(argv[2]);
    int C=atoi(argv[3]);
    int K=C;
    int blockSize = 256;
    int gridSize = 0;
    float time_gpu;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int L = H - (K - 1);
    int M = W - (K - 1);
    int KERNELS_NUM = L * M * C;

    int countA = H*W*C;
    const size_t sizeA = countA*sizeof(float);

    int radiusF = (K - 1) / 2;
    int countF = K*K*C;
    int countLR = L * M;
    int countAc = countF * countLR;
    const size_t sizeAc = countAc*sizeof(float);

    float *matA = (float *)malloc(sizeA);
    srand((unsigned)time(0));
    for (int i = 0; i < countA; i++) {
        matA[i] = rand()%10;
    }
    float *devA, *devAc, *retAc;
    hipMalloc((void**)&devA, sizeA); 
    hipMalloc((void**)&devAc, sizeAc); 
    retAc = (float *)malloc(sizeAc);
    hipMemcpy(devA, matA, sizeA, hipMemcpyHostToDevice); 

    if (gridSize == 0)
        gridSize = (KERNELS_NUM + blockSize - 1) / blockSize;
    
    hipEventRecord(start, 0);
    im2colOnDevice<<<gridSize, blockSize>>>(KERNELS_NUM, devAc, devA, radiusF, countF, L, M, K, C,H);    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_gpu, start, stop);
    hipMemcpy(retAc, devAc, sizeAc, hipMemcpyDeviceToHost);
    printf("共用时间%f ms",time_gpu);

    hipFree(devA);
    hipFree(devAc);
    free(matA);
    free(retAc);
    return 0;
}