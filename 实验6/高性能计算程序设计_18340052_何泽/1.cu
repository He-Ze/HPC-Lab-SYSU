#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define RUN_CPU 0

 __global__ void gemm_gpu(double *a,double *b, double *c, int m, int n, int k)
 { 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double tmp = 0;
    if( col < k && row < m) {
        for(int i = 0; i < n; i++) {
            tmp += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = tmp;
    }
} 

void gemm_cpu(double *host_a, double *host_b, double *host_result, int m, int n, int k) 
{
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            double tmp = 0.0;
            for (int h = 0; h < n; h++) {
                tmp += host_a[i * n + h] * host_b[h * k + j];
            }
            host_result[i * k + j] = tmp;
        }
    }
}
 
int main(int argc, char const *argv[])
{
    int block_size=atoi(argv[1]);
    int m=atoi(argv[2]);
    int n=atoi(argv[3]);
    int k=atoi(argv[4]);

    double *host_a, *host_b, *host_c_gpu, *host_c_cpu;
    double *device_a, *device_b, *device_c;
    float time_gpu, time_cpu;

    hipHostMalloc((void **) &host_a, sizeof(double)*m*n);
    hipHostMalloc((void **) &host_b, sizeof(double)*n*k);
    hipHostMalloc((void **) &host_c_gpu, sizeof(double)*m*k);
    hipHostMalloc((void **) &host_c_cpu, sizeof(double)*m*k);

    srand((unsigned)time(0));
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            host_a[i * n + j] = (double)rand() / (double)(RAND_MAX)*1e4;
        }
    }
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            host_b[i * k + j] = (double)rand() / (double)(RAND_MAX)*1e4;
        }
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMalloc((void **) &device_a, sizeof(double)*m*n);
    hipMalloc((void **) &device_b, sizeof(double)*n*k);
    hipMalloc((void **) &device_c, sizeof(double)*m*k);
    hipMemcpy(device_a, host_a, sizeof(double)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, sizeof(double)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + block_size - 1) / block_size;
    unsigned int grid_cols = (k + block_size - 1) / block_size;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);
    gemm_gpu<<<dimGrid, dimBlock>>>(device_a, device_b, device_c, m, n, k);

    hipMemcpy(host_c_gpu, device_c, sizeof(double)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_gpu, start, stop);
    printf("矩阵A维度%dx%d，矩阵B维度%dx%d，Block_size为%d，在GPU上运行时间: %f ms.\n", m, n, n, k, block_size, time_gpu);

    if(RUN_CPU){
        hipEventRecord(start, 0);
        gemm_cpu(host_a, host_b, host_c_cpu, m, n, k);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_cpu, start, stop);
        printf("矩阵A维度%dx%d，矩阵B维度%dx%d，Block_size为%d，在CPU上运行时间: %f ms.\n\n", m, n, n, k, block_size, time_cpu);
        int all_ok = 1;
        for (int i = 0; i < m; ++i){
            for (int j = 0; j < k; ++j){
                if(host_c_cpu[i*k + j] != host_c_gpu[i*k + j]){
                    all_ok = 0;
                }
            }
        }
        if(all_ok){
            printf("结果正确，加速比为%f\n", time_cpu / time_gpu);
        }
        else{
            printf("结果错误\n");
        }
    }
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c_gpu);
    hipHostFree(host_c_cpu);
    return 0;
}